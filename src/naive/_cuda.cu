#include "_cuda.h"


static hipMemcpyKind map_memcpy_kind(cuda_memcpy_kind_t kind) {
    switch (kind) {
        case cuda_memcpy_host_to_host: return hipMemcpyHostToHost;
        case cuda_memcpy_host_to_device: return hipMemcpyHostToDevice;
        case cuda_memcpy_device_to_host: return hipMemcpyDeviceToHost;
        case cuda_memcpy_device_to_device: return hipMemcpyDeviceToDevice;
        default: return hipMemcpyDefault;
    }
}


void* cuda_malloc(size_t size)
{
    void* dev_ptr = NULL;

    hipError_t err = hipMalloc(&dev_ptr, size);
    if (err != hipSuccess) {
        dev_ptr = NULL;
    }

    return dev_ptr;
}


uint32_t cuda_memcpy(void* to, void* from, size_t count, cuda_memcpy_kind_t kind)
{
    return hipMemcpy(to, from, count, map_memcpy_kind(kind)) != hipSuccess;
}


uint32_t cuda_memset(void* data, int val, size_t count)
{
    return hipMemset(data, val, count) != hipSuccess;
}


uint32_t cuda_free(void* ptr)
{
    return hipFree(ptr) != hipSuccess;
}
