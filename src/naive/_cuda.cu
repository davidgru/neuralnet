#include "hip/hip_runtime.h"
#include "_cuda.h"

extern "C" {
#include "log.h"
}

static hipMemcpyKind map_memcpy_kind(cuda_memcpy_kind_t kind) {
    switch (kind) {
        case cuda_memcpy_host_to_host: return hipMemcpyHostToHost;
        case cuda_memcpy_host_to_device: return hipMemcpyHostToDevice;
        case cuda_memcpy_device_to_host: return hipMemcpyDeviceToHost;
        case cuda_memcpy_device_to_device: return hipMemcpyDeviceToDevice;
        default: return hipMemcpyDefault;
    }
}


static bool cuda_props_init;
static cuda_props_t cuda_props;


const cuda_props_t* get_cuda_props()
{
    if (cuda_props_init) {
        return &cuda_props;
    }

    /* TODO check cuda device props */

    cuda_props.default_block_size_1d.x = 256;
    cuda_props.default_block_size_1d.y = 1;
    cuda_props.default_block_size_1d.z = 1;

    cuda_props.default_block_size_2d.x = 16;
    cuda_props.default_block_size_2d.y = 16;
    cuda_props.default_block_size_2d.z = 1;

    return &cuda_props;
}


uint32_t cuda_check_error(hipError_t error, const char* file, int line, bool abort)
{
    if (error != hipSuccess) {
        LOG_ERROR("%s:%d cuda error: %s\n", file, line, hipGetErrorString(error));
        if (abort) exit(error);
        return 1;
    }
    return 0;
}


void* cuda_malloc(size_t size)
{
    void* dev_ptr = NULL;

    hipError_t err = hipMalloc(&dev_ptr, size);
    CUDA_CHECK_ERROR(err);
    if (err != hipSuccess) {
        dev_ptr = NULL;
    }

    return dev_ptr;
}


uint32_t cuda_memcpy(void* to, const void* from, size_t count, cuda_memcpy_kind_t kind)
{
    CUDA_CHECK_ERROR(hipMemcpy(to, from, count, map_memcpy_kind(kind)));
    return 0;
}


uint32_t cuda_memset(void* data, int val, size_t count)
{
    CUDA_CHECK_ERROR(hipMemset(data, val, count));
    return 0;
}


uint32_t cuda_free(void* ptr)
{
    CUDA_CHECK_ERROR(hipFree(ptr));
    return 0;
}
