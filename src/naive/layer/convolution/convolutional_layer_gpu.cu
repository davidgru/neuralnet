#include "hip/hip_runtime.h"
#include "_cuda.h"

#include "convolutional_layer_internal.h"
extern "C" {
#include "tensor/tensor_math.h"
}


__global__
void conv2d_kernel(const float* input, const float* kernel, float* output, int input_height,
    int input_width, int output_height, int output_width, int kernel_height, int kernel_width,
    int stride_y, int stride_x, int padding_y, int padding_x, int dilation_y, int dilation_x,
    int skip_output_y, int skip_output_x, int flip_kernel)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int r = idx / output_width;
    const int c = idx % output_width;

    if (r < output_height && c < output_width) {        
        const int data_r = (r + skip_output_y) * stride_y - padding_y;
        const int data_c = (c + skip_output_x) * stride_x - padding_x;

        /* calculate the bounds of the kernel to skip the elements that are in the padding */
        const int kr_start = max(0, div_ceil(-data_r, dilation_y));
        const int kr_end = min(kernel_height, div_ceil(input_height - data_r, dilation_y));
        const int kc_start = max(0, div_ceil(-data_c, dilation_x));
        const int kc_end = min(kernel_width, div_ceil(input_width - data_c, dilation_x));

        float sum = 0.0;
        for (int kr = kr_start; kr < kr_end; kr++) { 
            for (int kc = kc_start; kc < kc_end; kc++) {
                const int data_rk = data_r + kr * dilation_y;
                const int data_ck = data_c + kc * dilation_x;
                const int kernel_idx = flip_kernel ?
                    ((kernel_height - kr - 1) * kernel_width + (kernel_width - kc - 1)) :
                    (kr * kernel_width + kc);
                sum += input[data_rk * input_width + data_ck]
                    * kernel[kernel_idx];
            }
        }
        output[r * output_width + c] += sum;
    }
}


__global__
void convolution_forward_kernel(const float* input, const float* filter, const float* bias, float* output,
    int batch_size, int output_channels, int input_channels, int input_height, int input_width,
    int filter_height, int filter_width, int output_height, int output_width, int stride_y, int stride_x,
    int padding_y, int padding_x, int dilation_y, int dilation_x, int skip_output_y, int skip_output_x,
    int flip_kernel)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int column = batch_idx % output_width; batch_idx /= output_width;
    const int row = batch_idx % output_height; batch_idx /= output_height;
    const int oc_idx = batch_idx % output_channels; batch_idx /= output_channels;

    const int input_row = (row + skip_output_y) * stride_y - padding_y;
    const int input_column = (column + skip_output_x) * stride_x - padding_x;

    input += batch_idx * input_channels * input_height * input_width;
    filter += oc_idx * input_channels * filter_height * filter_width;
    output += (batch_idx * output_channels + oc_idx) * output_height * output_width;

    if (batch_idx < batch_size) {
        float sum = 0.0f;
        for (int ic_idx = 0; ic_idx < input_channels; ic_idx++) {
            for (int kr = 0; kr < filter_height; kr++) {
                for (int kc = 0; kc < filter_width; kc++) {
                    const int data_rk = input_row + kr * dilation_y;
                    const int data_ck = input_column + kc * dilation_x;
                    const int kernel_idx = flip_kernel ?
                        ((filter_height - kr - 1) * filter_width + (filter_width - kc - 1)) :
                        (kr * filter_width + kc);
                    if (data_rk >= 0 && data_rk < input_height && data_ck >= 0 && data_ck < input_width) {
                        sum += input[data_rk * input_width + data_ck] * filter[kernel_idx];
                    }
                }
            }
            input += input_height * input_width;
            filter += filter_height * filter_width;
        }
        output[row * output_width + column] += sum + bias[oc_idx];
    }
}


__global__
void convolution_backward_data_kernel(const float* prev_grad, const float* filter, float* grad,
    int batch_size, int prev_grad_height, int prev_grad_width, int in_channels, int out_channels,
    int filter_height, int filter_width, int grad_height, int grad_width, int stride_y, int stride_x,
    int padding_y, int padding_x)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int column = batch_idx % grad_width; batch_idx /= grad_width;
    const int row = batch_idx % grad_height; batch_idx /= grad_height;
    const int ic_idx = batch_idx % out_channels; batch_idx /= out_channels;

    const int prev_grad_row = row + padding_y - filter_height + 1;
    const int prev_grad_column = column + padding_x - filter_width + 1;

    prev_grad += batch_idx * out_channels * prev_grad_height * prev_grad_width;
    filter += ic_idx * filter_height * filter_width;
    grad += (batch_idx * in_channels + ic_idx) * grad_height * grad_width;
    if (batch_idx < batch_size) {
        float sum = 0.0f;
        for (int oc_idx = 0; oc_idx < out_channels; oc_idx++) {
            for (int kr = 0; kr < filter_height; kr++) {
                for (int kc = 0; kc < filter_width; kc++) {
                    const int prev_grad_rk = prev_grad_row + kr * stride_y;
                    const int prev_grad_ck = prev_grad_column + kc * stride_x;
                    const int kernel_idx = (filter_height - kr - 1) * filter_width
                                            + (filter_width - kc - 1);
                    if (prev_grad_rk >= 0 && prev_grad_rk < prev_grad_height
                        && prev_grad_ck >= 0 && prev_grad_ck < prev_grad_width) {
                        sum += prev_grad[prev_grad_rk * prev_grad_width + prev_grad_ck]
                                * filter[kernel_idx];
                    }
                }
            }
            prev_grad += prev_grad_height * prev_grad_width;
            filter += filter_height * filter_width;
        }
        grad[row * grad_width + column] += sum;
    }
}


__global__
void convolution_backward_weights_kernel_parallel_batch(const float* input, const float* prev_grad, float* d_filter,
    int batch_size, int in_channels, int input_height, int input_width, int prev_grad_channels,
    int prev_grad_height, int prev_grad_width, int filter_height, int filter_width, int stride_y,
    int stride_x, int padding_y, int padding_x)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int kcol = batch_idx % filter_width; batch_idx /= filter_width;
    const int krow = batch_idx % filter_height; batch_idx /= filter_height;
    const int ic_idx = batch_idx % in_channels; batch_idx /= in_channels;
    const int oc_idx = batch_idx % prev_grad_channels; batch_idx /= prev_grad_channels;

    const int input_row = krow - padding_y;
    const int input_column = kcol - padding_x;        

    input += (batch_idx * in_channels + ic_idx) * input_height * input_width;
    prev_grad += (batch_idx * prev_grad_channels + oc_idx) * prev_grad_height * prev_grad_width;
    d_filter += ((batch_idx * prev_grad_channels + oc_idx) * in_channels + ic_idx) * filter_height * filter_width;

    if (batch_idx < batch_size) {
        float sum = 0.0f;
        for (int gr = 0; gr < prev_grad_height; gr++) {
            for (int gc = 0; gc < prev_grad_width; gc++) {
                const int data_rk = input_row + gr * stride_y;
                const int data_ck = input_column + gc * stride_x;
                if (data_rk >= 0 && data_rk < input_height
                    && data_ck >= 0 && data_ck < input_width) {
                    sum += input[data_rk * input_width + data_ck]
                         * prev_grad[gr * prev_grad_width + gc];
                }
            }
        }
        d_filter[krow * filter_width + kcol] += sum;
    }
}


void conv2d_gpu(const float* input, const float* kernel, float* output, int32_t input_height,
    int32_t input_width, int32_t kernel_height, int32_t kernel_width, int32_t stride_y,
    int32_t stride_x, int32_t padding_y, int32_t padding_x, int32_t dilation_y, int32_t dilation_x,
    int32_t skip_output_y, int32_t skip_output_x, int32_t flip_kernel)
{
    const int output_height = conv_output_size(input_height, kernel_height, stride_y, dilation_y,
        padding_y) - 2 * skip_output_y;
    const int output_width = conv_output_size(input_width, kernel_width, stride_x, dilation_x,
        padding_x) - 2 * skip_output_x;

    const cuda_props_t* props = get_cuda_props();
    const int block_size = props->default_block_size_1d.x;
    const int num_blocks = cuda_calc_num_blocks((output_height * output_width), block_size);

    conv2d_kernel<<<num_blocks, block_size>>>(input, kernel, output, input_height, input_width, output_height,
        output_width, kernel_height, kernel_width, stride_y, stride_x, padding_y, padding_x, dilation_y,
        dilation_x, skip_output_y, skip_output_x, flip_kernel);
    CUDA_CHECK_LAST_ERROR();
}


void convolution_forward_gpu(const tensor_t* input, const tensor_t* filter, const tensor_t* bias,
    tensor_t* output, int32_t stride_y, int32_t stride_x, int32_t padding_y, int32_t padding_x,
    int32_t dilation_y, int32_t dilation_x, int32_t skip_output_y, int32_t skip_output_x,
    int32_t flip_kernel)
{
    const unsigned int num_threads = tensor_batch_size(output) * tensor_channels(output)
        * tensor_height(output) * tensor_width(output);

    const cuda_props_t* props = get_cuda_props();
    const dim3 block_size = props->default_block_size_1d;
    const dim3 block_dim = {
        cuda_calc_num_blocks(num_threads, block_size.x), 1, 1
    };

    convolution_forward_kernel<<<block_dim, block_size>>>(input->data, filter->data, bias->data, output->data,
        tensor_batch_size(input), tensor_channels(output), tensor_channels(input), tensor_height(input),
        tensor_width(input), _filter_height(filter), _filter_width(filter), tensor_height(output), tensor_width(output),
        stride_y, stride_x, padding_y, padding_x, dilation_y, dilation_x, skip_output_y, skip_output_x, flip_kernel);
    CUDA_CHECK_LAST_ERROR();
}


void convolution_backward_data_gpu(const tensor_t* prev_grad, const tensor_t* filter, tensor_t* grad,
    int32_t stride_y, int32_t stride_x, int32_t padding_y, int32_t padding_x)
{
    const unsigned int num_threads = tensor_batch_size(grad) * tensor_channels(grad)
        * tensor_height(grad) * tensor_width(grad);

    const cuda_props_t* props = get_cuda_props();
    const dim3 block_size = props->default_block_size_1d;
    const dim3 block_dim = {
        cuda_calc_num_blocks(num_threads, block_size.x), 1, 1
    };

    convolution_backward_data_kernel<<<block_dim, block_size>>>(prev_grad->data, filter->data, grad->data,
        tensor_batch_size(prev_grad), tensor_height(prev_grad), tensor_width(prev_grad), tensor_channels(grad),
        tensor_channels(prev_grad), _filter_height(filter), _filter_width(filter), tensor_height(grad),
        tensor_width(grad), stride_y, stride_x, padding_y, padding_x);
    CUDA_CHECK_LAST_ERROR();
}


void convolution_backward_weights_gpu(const tensor_t* input, const tensor_t* prev_grad, tensor_t* d_weights,
    tensor_t* d_bias, int32_t stride_y, int32_t stride_x, int32_t padding_y, int32_t padding_x)
{
    tensor_t d_weights_per_batch;
    tensor_shape_t d_weights_per_batch_shape = make_tensor_shape(5,
        tensor_batch_size(input),
        tensor_channels(prev_grad),
        tensor_channels(input),
        _filter_height(d_weights),
        _filter_width(d_weights)
    );
    tensor_allocate_device(&d_weights_per_batch, &d_weights_per_batch_shape, device_gpu);
    tensor_set_zero(&d_weights_per_batch);

    const unsigned int d_weights_num_threads = tensor_batch_size(input) * tensor_channels(prev_grad) * tensor_channels(input) 
        * _filter_height(d_weights) * _filter_width(d_weights);

    const cuda_props_t* props = get_cuda_props();
    const dim3 block_size = props->default_block_size_1d;
    const dim3 block_dim = {
        cuda_calc_num_blocks(d_weights_num_threads, block_size.x), 1, 1
    };

    /* compute d_weights for each batch in parallel */
    convolution_backward_weights_kernel_parallel_batch<<<block_dim, block_size>>>(input->data, prev_grad->data,
        d_weights_per_batch.data, tensor_batch_size(input), tensor_channels(input), tensor_height(input), tensor_width(input),
        tensor_channels(prev_grad), tensor_height(prev_grad), tensor_width(prev_grad), _filter_height(d_weights),
        _filter_width(d_weights), stride_y, stride_x, padding_y, padding_x);
    CUDA_CHECK_LAST_ERROR();

    /* sum d_weights along batch dimension */
    tensor_sum_axis(d_weights, &d_weights_per_batch, 0);

    /* d_bias is simply a sum of the previous gradient */
    /* for lack of better suiting function, first reduce width and height dim and then reduce batch dim */
    tensor_t prev_grad_view_flattened_wh = {
        .shape = make_tensor_shape(3,
            tensor_batch_size(prev_grad),
            tensor_channels(prev_grad),
            tensor_per_channel_size(prev_grad)
        ),
        .device = device_gpu,
        .data = prev_grad->data
    };

    /* reduce height and width */
    tensor_t d_bias_per_batch;
    tensor_shape_t prev_reduced_wh_shape = make_tensor_shape(2,
        tensor_batch_size(prev_grad),
        tensor_channels(prev_grad)
    );
    tensor_allocate_device(&d_bias_per_batch, &prev_reduced_wh_shape, device_gpu);
    tensor_set_zero(&d_bias_per_batch);

    tensor_sum_axis(&d_bias_per_batch, &prev_grad_view_flattened_wh, 2);
    /* reduce batch dimension */
    tensor_sum_axis(d_bias, &d_bias_per_batch, 0);

    tensor_destory(&d_weights_per_batch);
    tensor_destory(&d_bias_per_batch);
}


#if 0 /* slower because batch is handled sequentially but left for reference */
__global__
void convolution_backward_weights_kernel(const float* input, const float* prev_grad, float* d_filter,
    int batch_size, int in_channels, int input_height, int input_width, int prev_grad_channels,
    int prev_grad_height, int prev_grad_width, int filter_height, int filter_width, int stride_y,
    int stride_x, int padding_y, int padding_x)
{
    int oc_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int kcol = oc_idx % filter_width; oc_idx /= filter_width;
    const int krow = oc_idx % filter_height; oc_idx /= filter_height;
    const int ic_idx = oc_idx % in_channels; oc_idx /= in_channels;

    const int input_row = krow - padding_y;
    const int input_column = kcol - padding_x;        

    input += ic_idx * input_height * input_width;
    prev_grad += oc_idx * prev_grad_height * prev_grad_width;
    d_filter += (oc_idx * in_channels + ic_idx) * filter_height * filter_width;

    if (oc_idx < prev_grad_channels) {
        float sum = 0.0f;
        for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
            for (int gr = 0; gr < prev_grad_height; gr++) {
                for (int gc = 0; gc < prev_grad_width; gc++) {
                    const int data_rk = input_row + gr * stride_y;
                    const int data_ck = input_column + gc * stride_x;
                    if (data_rk >= 0 && data_rk < input_height
                        && data_ck >= 0 && data_ck < input_width) {
                        sum += input[data_rk * input_width + data_ck]
                             * prev_grad[gr * prev_grad_width + gc];
                    }
                }
            }
            input += in_channels * input_height * input_width;
            prev_grad += prev_grad_channels * prev_grad_height * prev_grad_width;
        }
        d_filter[krow * filter_width + kcol] += sum;
    }
}


__global__
void convolution_backward_bias_kernel(const float* prev_grad, float* d_bias, int batch_size,
    int prev_grad_channels, int prev_grad_height, int prev_grad_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < prev_grad_channels) {
        float sum = 0.0f;
        for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
            for (int gr = 0; gr < prev_grad_height; gr++) {
                for (int gc = 0; gc < prev_grad_width; gc++) {
                    sum += prev_grad[((batch_idx * prev_grad_channels + idx)
                            * prev_grad_height + gr) * prev_grad_width + gc];
                }
            }
        }
        d_bias[idx] += sum;
    }
}


void convolution_backward_weights_gpu(const tensor_t* input, const tensor_t* prev_grad, tensor_t* d_weights,
    tensor_t* d_bias, int32_t stride_y, int32_t stride_x, int32_t padding_y, int32_t padding_x)
{
    const unsigned int num_threads = tensor_channels(prev_grad) * tensor_channels(d_weights) 
        * _filter_height(d_weights) * _filter_width(d_weights);

    const cuda_props_t* props = get_cuda_props();
    const dim3 block_size = props->default_block_size_1d;
    const dim3 block_dim = {
        cuda_calc_num_blocks(num_threads, block_size.x), 1, 1
    };

    convolution_backward_weights_kernel<<<block_dim, block_size>>>(input->data, prev_grad->data, d_weights->data,
        tensor_batch_size(input), tensor_channels(input), tensor_height(input), tensor_width(input),
        tensor_channels(prev_grad), tensor_height(prev_grad), tensor_width(prev_grad), _filter_height(d_weights),
        _filter_width(d_weights), stride_y, stride_x, padding_y, padding_x);
    CUDA_CHECK_LAST_ERROR();

    const dim3 bias_bs = props->default_block_size_1d;
    const dim3 bias_nb = {
        cuda_calc_num_blocks(tensor_channels(prev_grad), bias_bs.x)
    };

    convolution_backward_bias_kernel<<<bias_nb, bias_nb>>>(prev_grad->data, d_bias->data, tensor_batch_size(prev_grad),
        tensor_channels(prev_grad), tensor_height(prev_grad), tensor_width(prev_grad));
    CUDA_CHECK_LAST_ERROR();
}
#endif
